/*
!pip install git+https://github.com/andreinechaev/nvcc4jupyter.git
%load_ext nvcc_plugin
*/

// %%cu

#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
using namespace std;

const int total = 9;
const int row1 = 4;
const int col1 = 5;
const int col2 = 3;

__global__ void matrixMultiplication(int *mat1, int *mat2, int *ans)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < row1 && col < col2)
    {
        int sum = 0;
        for (int i = 0; i < col1; ++i)
        {
            sum += mat1[row * col1 + i] + mat2[i * col2 + col];
        }
        ans[row * col2 + col] = sum;
    }
}

void generateRandomMatrix(int *matrix, int row, int col)
{
    for (int r = 0; r < row; ++r)
    {
        for (int c = 0; c < col; ++c)
        {
            matrix[r * col + c] = rand() % 10;
        }
    }
}

void printMatrix(int *matrix, int row, int col, int id)
{

    cout << "Result[" << id << "] =\n";
    for (int r = 0; r < row; ++r)
    {
        for (int c = 0; c < col; ++c)
        {
            cout << matrix[r * col + c] << "\t";
        }
        cout << endl;
    }
    cout << "\n\n";
}

int main()
{
    // Host (CPU) matrix
    int *hMat1;
    int *hMat2;
    int *hAns;

    // Host memory allocation
    hMat1 = (int *)malloc(total * row1 * col1 * sizeof(int));
    hMat2 = (int *)malloc(total * col1 * col2 * sizeof(int));
    hAns = (int *)malloc(total * row1 * col2 * sizeof(int));

    srand(time(nullptr));

    // Random Matrix Generation
    for (int i = 0; i < total; ++i)
    {
        generateRandomMatrix(hMat1 + i * row1 * col1, row1, col1);
        generateRandomMatrix(hMat2 + i * col1 * col2, col1, col2);
    }

    // Device (GPU) matrix
    int *dMat1;
    int *dMat2;
    int *dAns;

    // Device memory allocation
    hipMalloc(&dMat1, total * row1 * col1 * sizeof(int));
    hipMalloc(&dMat2, total * col1 * col2 * sizeof(int));
    hipMalloc(&dAns, total * row1 * col2 * sizeof(int));

    hipEvent_t startTime;
    hipEventCreate(&startTime);

    hipEvent_t endTime;
    hipEventCreate(&endTime);

    // Copy data from Host to Device
    hipMemcpy(dMat1, hMat1, total * row1 * col1 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dMat2, hMat2, total * col1 * col2 * sizeof(int), hipMemcpyHostToDevice);

    // Define grid dimension & block dimension of thread
    dim3 blockDim(16, 16);
    dim3 gridDim((col2 + blockDim.x - 1) / blockDim.x, (row1 + blockDim.y) / blockDim.y);

    hipEventRecord(startTime);

    // Matrix multiplication kernel
    for (int i = 0; i < total; ++i)
    {
        matrixMultiplication<<<gridDim, blockDim>>>(dMat1 + i * row1 * col1, dMat2 + i * col1 * col2, dAns + i * row1 * col2);
    }

    hipEventRecord(endTime);
    hipEventSynchronize(endTime);

    // Copy the result back to the host
    hipMemcpy(hAns, dAns, total * row1 * col2 * sizeof(int), hipMemcpyDeviceToHost);

    float timeTaken = 0;
    hipEventElapsedTime(&timeTaken, startTime, endTime);
    cout << "Time taken  to execute the Matrix Multiplication: " << timeTaken << "ms\n\n";

    //... Print the result
    for (int i = 0; i < total; i++)
    {
        printMatrix(hAns + i * row1 * col2, row1, col2, i);
    }

    hipFree(dMat1);
    hipFree(dMat2);
    hipFree(dAns);
    free(hMat1);
    free(hMat2);
    free(hAns);

    return 0;
}
